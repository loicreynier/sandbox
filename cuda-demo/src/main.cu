
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N (2048 * 2028)
#define MAX_ERR 1e-6

__global__ void VecAdd(float *a, float *b, float *c, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n)
    c[i] = a[i] + b[i];
}

void fill_array_random_floats(float *array, float size, float min, float max) {
  srand(time(NULL));

  for (int i = 0; i < size; i++) {
    array[i] = min + ((float)rand() / RAND_MAX) * (max - min);
  }
}

int main(void) {
  float *h_a, *h_b, *h_c;
  float *d_a, *d_b, *d_c;
  size_t size = N * sizeof(float);

  // Allocate host memory
  h_a = (float *)malloc(size);
  h_b = (float *)malloc(size);
  h_c = (float *)malloc(size);

  // Allocate device memory
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  // Initialize input vectors
  fill_array_random_floats(h_b, N, 0.0f, 7.0f);
  fill_array_random_floats(h_a, N, 11.0f, 17.0f);

  // Copy vectors from host memory to device memory
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

  // Invoke kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

  // Copy result from device memory to host memory
  hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

  // Synchronize before running more host code
  hipDeviceSynchronize();

  // Verification
  for (int i = 0; i < N; i++) {
    assert(fabs(h_c[i] - h_a[i] - h_b[i]) < MAX_ERR);
  }

  // Deallocate host memory
  free(h_a);
  free(h_b);
  free(h_c);

  // Deallocate device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
